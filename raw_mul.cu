#include "hip/hip_runtime.h"
#include "gplex_mul.h"
#include "hipblas.h"

__global__ void raw_naive_mult_kn(const float* RESTRICT a,
    const float* RESTRICT b, float* RESTRICT c, const int N)
{
  for (int n = threadIdx.x + blockIdx.x * blockDim.x;
       n < N;
       n += blockDim.x * gridDim.x) {

    for (int i = 0; i < 6; ++i) {
      for (int j = 0; j < 6; ++j) {
        for (int k = 0; k < 6; ++k) {
          c[n + N*(i + 6*j)] += a[n + N*(i + 6*k)] * b[n + N*(k + 6*j)];
        }
      }
    }
  }
}

__global__ void raw_reg_c_mult_kn(const float* RESTRICT a, const float* RESTRICT b, 
    float* c, const int N)
{
  for (int n = threadIdx.x + blockIdx.x * blockDim.x;
       n < N;
       n += blockDim.x * gridDim.x) {

    for (int i = 0; i < 6; ++i) {
      for (int j = 0; j < 6; ++j) {
        float c_tmp = 0;
        for (int k = 0; k < 6; ++k) {
          c_tmp += a[n + N*(i + 6*k)] * b[n + N*(k + 6*j)];
        }
        c[n + N*(i + 6*j)] = c_tmp;
      }
    }
  }
}

/* Try to do all from the registers; the code is adjusted to get to 100% FLOP efficiency */
__global__ void raw_regall_c_mult_loop_kn(const float* RESTRICT a, const float* RESTRICT b,
                                          float* c, const int N, const int nN)
{

  float a_ar[36];
  float b_ar[36];
  float c_ar[36];
  for (int n = threadIdx.x + blockIdx.x * blockDim.x;
       n < N;
       n += blockDim.x * gridDim.x) {
    for (int i = 0; i < 36; ++i){
      const int idx = n + N*i;
      a_ar[i] = a[idx];
      b_ar[i] = b[idx];
      c_ar[i] = 0.f;
    }
  }

  for (int n = threadIdx.x + blockIdx.x * blockDim.x;
       n < N;
       n += blockDim.x * gridDim.x) {

    for (int oLoop = 0; oLoop< nN; ++oLoop){

      for (int i = 0; i < 6; ++i) {
        for (int j = 0; j < 6; ++j) {
          float c_tmp = 0.f;
          for (int k = 0; k < 6; ++k) {
            c_tmp += a_ar[i + 6*k] * b_ar[k + 6*j];
          }
          c_ar[i + 6*j] += c_tmp*3.4f;
        }
      }
    }
  }//oLoop< nN; ++oLoop){

  for (int n = threadIdx.x + blockIdx.x * blockDim.x;
       n < N;
       n += blockDim.x * gridDim.x) {
    for (int i = 0; i < 36; ++i){
      const int idx = n + N*i;
      c[idx] = c_ar[i];
    }
  }

}


__global__ void raw_reg_c_mult_loop_kn(const float* RESTRICT const a, const float* RESTRICT const b, 
    float* c, const int N, const int nN)
{

  for (int oLoop = 0; oLoop< nN; ++oLoop){
    for (int n = threadIdx.x + blockIdx.x * blockDim.x;
         n < N;
         n += blockDim.x * gridDim.x) {
      
      float a_ar[36];
      float b_ar[36];
      for (int i = 0; i < 36; ++i){
        const int idx = n + N*i;
        a_ar[i] = a[idx];
        b_ar[i] = b[idx];
      }
      for (int i = 0; i < 6; ++i) {
        for (int j = 0; j < 6; ++j) {
          float c_tmp = 0.f;
          for (int k = 0; k < 6; ++k) {
            c_tmp += a_ar[i + 6*k] * b_ar[k + 6*j];
          }
          c[n + N*(i + 6*j)] = c_tmp;
        }
      }
    }
  }//oLoop< nN; ++oLoop){
}

/** like raw_reg_c_mult_loop_kn but with vectorized loads and 
    therefore a different memory layout of the matrix */
__global__ void raw_reg_c_mult_loop_kn_vl(const float* RESTRICT const a, const float* RESTRICT const b, 
    float* c, const int N, const int nN)
{
  for (int oLoop = 0; oLoop< nN; ++oLoop){
    for (int n = threadIdx.x + blockIdx.x * blockDim.x;
         n < N;
         n += blockDim.x * gridDim.x) {
      
      float a_ar[36];
      float b_ar[36];
      for (int i = 0; i < 9; ++i){
        const int idx = n + N*i;
        reinterpret_cast<float4*>(a_ar)[i] = reinterpret_cast<const float4*>(a)[idx];
        reinterpret_cast<float4*>(b_ar)[i] = reinterpret_cast<const float4*>(b)[idx];
      }
      
      for (int i = 0; i < 6; ++i) {
        for (int j = 0; j < 6; ++j) {
          float c_tmp = 0.f;
          for (int k = 0; k < 6; ++k) {
            c_tmp += a_ar[i + 6*k] * b_ar[k + 6*j];
          }
          c[n + N*(i + 6*j)] = c_tmp;
        }
      }
    }
  }//oLoop< nN; ++oLoop){
}

/** like raw_reg_c_mult_loop_kn_vl but also using 
    vectorized stores */
__global__ void raw_reg_c_mult_loop_kn_vl_vs(const float* RESTRICT const a, const float* RESTRICT const b, 
    float* c, const int N, const int nN)
{
  for (int oLoop = 0; oLoop< nN; ++oLoop){
    for (int n = threadIdx.x + blockIdx.x * blockDim.x;
         n < N;
         n += blockDim.x * gridDim.x) {
      
      float a_ar[36];
      float b_ar[36];
      float c_ar[36];
      for (int i = 0; i < 9; ++i){
        const int idx = n + N*i;
        reinterpret_cast<float4*>(a_ar)[i] = reinterpret_cast<const float4*>(a)[idx];
        reinterpret_cast<float4*>(b_ar)[i] = reinterpret_cast<const float4*>(b)[idx];
      }
      
      for (int i = 0; i < 6; ++i) {
        for (int j = 0; j < 6; ++j) {
          float c_tmp = 0.f;
          for (int k = 0; k < 6; ++k) {
            c_tmp += a_ar[i + 6*k] * b_ar[k + 6*j];
          }
	  c_ar[i + 6 * j] = c_tmp;
        }
      }

      // write back
      for (int i = 0; i < 9; ++i){
	const int idx = n + N*i;
	reinterpret_cast<float4*>(c)[idx] = reinterpret_cast<const float4*>(c_ar)[i];
      }
    }
  }//oLoop< nN; ++oLoop){
}


/** like raw_reg_c_mult_loop_kn_vl but assuming that the matrix B is 
    transposed thus making the index calculation for matrix A
    and B the same */
__global__ void raw_reg_c_mult_loop_kn_vl_transp(const float* RESTRICT const a, const float* RESTRICT const b, 
    float* c, const int N, const int nN)
{
  for (int oLoop = 0; oLoop< nN; ++oLoop){
    for (int n = threadIdx.x + blockIdx.x * blockDim.x;
         n < N;
         n += blockDim.x * gridDim.x) {
      
      float a_ar[36];
      float b_ar[36];
      for (int i = 0; i < 9; ++i){
        const int idx = n + N*i;
        reinterpret_cast<float4*>(a_ar)[i] = reinterpret_cast<const float4*>(a)[idx];
        reinterpret_cast<float4*>(b_ar)[i] = reinterpret_cast<const float4*>(b)[idx];
      }
      
      for (int i = 0; i < 6; ++i) {
        for (int j = 0; j < 6; ++j) {
          float c_tmp = 0.f;
          for (int k = 0; k < 36; k += 6) {
            c_tmp += a_ar[i + k] * b_ar[j + k];
          }
	  c[n + N *(i + 6 * j)] = c_tmp;
        }
      }
    }
  }//oLoop< nN; ++oLoop){
}


__global__ void raw_reg_c_mult_loop_unroll_kn(const float* RESTRICT const a, const float* RESTRICT const b, 
    float* c, const int N, const int nN)
{
  for (int oLoop = 0; oLoop< nN; ++oLoop){
    for (int n = threadIdx.x + blockIdx.x * blockDim.x;
         n < N;
         n += blockDim.x * gridDim.x) {
      float c_temp;
      float a_00 = a[n + N*0];
      float b_00 = b[n + N*0];
      c_temp =  a_00  * b_00;
      float a_01 = a[n + N*6];
      float b_01 = b[n + N*6];
      float a_10 = a[n + N*1];
      float b_10 = b[n + N*1];
      c_temp +=  a_01  * b_10;
      float a_02 = a[n + N*12];
      float b_02 = b[n + N*12];
      float a_20 = a[n + N*2];
      float b_20 = b[n + N*2];
      c_temp +=  a_02  * b_20;
      float a_03 = a[n + N*18];
      float b_03 = b[n + N*18];
      float a_30 = a[n + N*3];
      float b_30 = b[n + N*3];
      c_temp +=  a_03  * b_30;
      float a_04 = a[n + N*24];
      float b_04 = b[n + N*24];
      float a_40 = a[n + N*4];
      float b_40 = b[n + N*4];
      c_temp +=  a_04  * b_40;
      float a_05 = a[n + N*30];
      float b_05 = b[n + N*30];
      float a_50 = a[n + N*5];
      float b_50 = b[n + N*5];
      c_temp +=  a_05  * b_50;
      c[n + N*0 ] = c_temp;
      c_temp =  a_00  * b_01;
      float a_11 = a[n + N*7];
      float b_11 = b[n + N*7];
      c_temp +=  a_01  * b_11;
      float a_21 = a[n + N*8];
      float b_21 = b[n + N*8];
      c_temp +=  a_02  * b_21;
      float a_31 = a[n + N*9];
      float b_31 = b[n + N*9];
      c_temp +=  a_03  * b_31;
      float a_41 = a[n + N*10];
      float b_41 = b[n + N*10];
      c_temp +=  a_04  * b_41;
      float a_51 = a[n + N*11];
      float b_51 = b[n + N*11];
      c_temp +=  a_05  * b_51;
      c[n + N*6 ] = c_temp;
      c_temp =  a_00  * b_02;
      float a_12 = a[n + N*13];
      float b_12 = b[n + N*13];
      c_temp +=  a_01  * b_12;
      float a_22 = a[n + N*14];
      float b_22 = b[n + N*14];
      c_temp +=  a_02  * b_22;
      float a_32 = a[n + N*15];
      float b_32 = b[n + N*15];
      c_temp +=  a_03  * b_32;
      float a_42 = a[n + N*16];
      float b_42 = b[n + N*16];
      c_temp +=  a_04  * b_42;
      float a_52 = a[n + N*17];
      float b_52 = b[n + N*17];
      c_temp +=  a_05  * b_52;
      c[n + N*12 ] = c_temp;
      c_temp =  a_00  * b_03;
      float a_13 = a[n + N*19];
      float b_13 = b[n + N*19];
      c_temp +=  a_01  * b_13;
      float a_23 = a[n + N*20];
      float b_23 = b[n + N*20];
      c_temp +=  a_02  * b_23;
      float a_33 = a[n + N*21];
      float b_33 = b[n + N*21];
      c_temp +=  a_03  * b_33;
      float a_43 = a[n + N*22];
      float b_43 = b[n + N*22];
      c_temp +=  a_04  * b_43;
      float a_53 = a[n + N*23];
      float b_53 = b[n + N*23];
      c_temp +=  a_05  * b_53;
      c[n + N*18 ] = c_temp;
      c_temp =  a_00  * b_04;
      float a_14 = a[n + N*25];
      float b_14 = b[n + N*25];
      c_temp +=  a_01  * b_14;
      float a_24 = a[n + N*26];
      float b_24 = b[n + N*26];
      c_temp +=  a_02  * b_24;
      float a_34 = a[n + N*27];
      float b_34 = b[n + N*27];
      c_temp +=  a_03  * b_34;
      float a_44 = a[n + N*28];
      float b_44 = b[n + N*28];
      c_temp +=  a_04  * b_44;
      float a_54 = a[n + N*29];
      float b_54 = b[n + N*29];
      c_temp +=  a_05  * b_54;
      c[n + N*24 ] = c_temp;
      c_temp =  a_00  * b_05;
      float a_15 = a[n + N*31];
      float b_15 = b[n + N*31];
      c_temp +=  a_01  * b_15;
      float a_25 = a[n + N*32];
      float b_25 = b[n + N*32];
      c_temp +=  a_02  * b_25;
      float a_35 = a[n + N*33];
      float b_35 = b[n + N*33];
      c_temp +=  a_03  * b_35;
      float a_45 = a[n + N*34];
      float b_45 = b[n + N*34];
      c_temp +=  a_04  * b_45;
      float a_55 = a[n + N*35];
      float b_55 = b[n + N*35];
      c_temp +=  a_05  * b_55;
      c[n + N*30 ] = c_temp;
      c_temp =  a_10  * b_00;
      c_temp +=  a_11  * b_10;
      c_temp +=  a_12  * b_20;
      c_temp +=  a_13  * b_30;
      c_temp +=  a_14  * b_40;
      c_temp +=  a_15  * b_50;
      c[n + N*1 ] = c_temp;
      c_temp =  a_10  * b_01;
      c_temp +=  a_11  * b_11;
      c_temp +=  a_12  * b_21;
      c_temp +=  a_13  * b_31;
      c_temp +=  a_14  * b_41;
      c_temp +=  a_15  * b_51;
      c[n + N*7 ] = c_temp;
      c_temp =  a_10  * b_02;
      c_temp +=  a_11  * b_12;
      c_temp +=  a_12  * b_22;
      c_temp +=  a_13  * b_32;
      c_temp +=  a_14  * b_42;
      c_temp +=  a_15  * b_52;
      c[n + N*13 ] = c_temp;
      c_temp =  a_10  * b_03;
      c_temp +=  a_11  * b_13;
      c_temp +=  a_12  * b_23;
      c_temp +=  a_13  * b_33;
      c_temp +=  a_14  * b_43;
      c_temp +=  a_15  * b_53;
      c[n + N*19 ] = c_temp;
      c_temp =  a_10  * b_04;
      c_temp +=  a_11  * b_14;
      c_temp +=  a_12  * b_24;
      c_temp +=  a_13  * b_34;
      c_temp +=  a_14  * b_44;
      c_temp +=  a_15  * b_54;
      c[n + N*25 ] = c_temp;
      c_temp =  a_10  * b_05;
      c_temp +=  a_11  * b_15;
      c_temp +=  a_12  * b_25;
      c_temp +=  a_13  * b_35;
      c_temp +=  a_14  * b_45;
      c_temp +=  a_15  * b_55;
      c[n + N*31 ] = c_temp;
      c_temp =  a_20  * b_00;
      c_temp +=  a_21  * b_10;
      c_temp +=  a_22  * b_20;
      c_temp +=  a_23  * b_30;
      c_temp +=  a_24  * b_40;
      c_temp +=  a_25  * b_50;
      c[n + N*2 ] = c_temp;
      c_temp =  a_20  * b_01;
      c_temp +=  a_21  * b_11;
      c_temp +=  a_22  * b_21;
      c_temp +=  a_23  * b_31;
      c_temp +=  a_24  * b_41;
      c_temp +=  a_25  * b_51;
      c[n + N*8 ] = c_temp;
      c_temp =  a_20  * b_02;
      c_temp +=  a_21  * b_12;
      c_temp +=  a_22  * b_22;
      c_temp +=  a_23  * b_32;
      c_temp +=  a_24  * b_42;
      c_temp +=  a_25  * b_52;
      c[n + N*14 ] = c_temp;
      c_temp =  a_20  * b_03;
      c_temp +=  a_21  * b_13;
      c_temp +=  a_22  * b_23;
      c_temp +=  a_23  * b_33;
      c_temp +=  a_24  * b_43;
      c_temp +=  a_25  * b_53;
      c[n + N*20 ] = c_temp;
      c_temp =  a_20  * b_04;
      c_temp +=  a_21  * b_14;
      c_temp +=  a_22  * b_24;
      c_temp +=  a_23  * b_34;
      c_temp +=  a_24  * b_44;
      c_temp +=  a_25  * b_54;
      c[n + N*26 ] = c_temp;
      c_temp =  a_20  * b_05;
      c_temp +=  a_21  * b_15;
      c_temp +=  a_22  * b_25;
      c_temp +=  a_23  * b_35;
      c_temp +=  a_24  * b_45;
      c_temp +=  a_25  * b_55;
      c[n + N*32 ] = c_temp;
      c_temp =  a_30  * b_00;
      c_temp +=  a_31  * b_10;
      c_temp +=  a_32  * b_20;
      c_temp +=  a_33  * b_30;
      c_temp +=  a_34  * b_40;
      c_temp +=  a_35  * b_50;
      c[n + N*3 ] = c_temp;
      c_temp =  a_30  * b_01;
      c_temp +=  a_31  * b_11;
      c_temp +=  a_32  * b_21;
      c_temp +=  a_33  * b_31;
      c_temp +=  a_34  * b_41;
      c_temp +=  a_35  * b_51;
      c[n + N*9 ] = c_temp;
      c_temp =  a_30  * b_02;
      c_temp +=  a_31  * b_12;
      c_temp +=  a_32  * b_22;
      c_temp +=  a_33  * b_32;
      c_temp +=  a_34  * b_42;
      c_temp +=  a_35  * b_52;
      c[n + N*15 ] = c_temp;
      c_temp =  a_30  * b_03;
      c_temp +=  a_31  * b_13;
      c_temp +=  a_32  * b_23;
      c_temp +=  a_33  * b_33;
      c_temp +=  a_34  * b_43;
      c_temp +=  a_35  * b_53;
      c[n + N*21 ] = c_temp;
      c_temp =  a_30  * b_04;
      c_temp +=  a_31  * b_14;
      c_temp +=  a_32  * b_24;
      c_temp +=  a_33  * b_34;
      c_temp +=  a_34  * b_44;
      c_temp +=  a_35  * b_54;
      c[n + N*27 ] = c_temp;
      c_temp =  a_30  * b_05;
      c_temp +=  a_31  * b_15;
      c_temp +=  a_32  * b_25;
      c_temp +=  a_33  * b_35;
      c_temp +=  a_34  * b_45;
      c_temp +=  a_35  * b_55;
      c[n + N*33 ] = c_temp;
      c_temp =  a_40  * b_00;
      c_temp +=  a_41  * b_10;
      c_temp +=  a_42  * b_20;
      c_temp +=  a_43  * b_30;
      c_temp +=  a_44  * b_40;
      c_temp +=  a_45  * b_50;
      c[n + N*4 ] = c_temp;
      c_temp =  a_40  * b_01;
      c_temp +=  a_41  * b_11;
      c_temp +=  a_42  * b_21;
      c_temp +=  a_43  * b_31;
      c_temp +=  a_44  * b_41;
      c_temp +=  a_45  * b_51;
      c[n + N*10 ] = c_temp;
      c_temp =  a_40  * b_02;
      c_temp +=  a_41  * b_12;
      c_temp +=  a_42  * b_22;
      c_temp +=  a_43  * b_32;
      c_temp +=  a_44  * b_42;
      c_temp +=  a_45  * b_52;
      c[n + N*16 ] = c_temp;
      c_temp =  a_40  * b_03;
      c_temp +=  a_41  * b_13;
      c_temp +=  a_42  * b_23;
      c_temp +=  a_43  * b_33;
      c_temp +=  a_44  * b_43;
      c_temp +=  a_45  * b_53;
      c[n + N*22 ] = c_temp;
      c_temp =  a_40  * b_04;
      c_temp +=  a_41  * b_14;
      c_temp +=  a_42  * b_24;
      c_temp +=  a_43  * b_34;
      c_temp +=  a_44  * b_44;
      c_temp +=  a_45  * b_54;
      c[n + N*28 ] = c_temp;
      c_temp =  a_40  * b_05;
      c_temp +=  a_41  * b_15;
      c_temp +=  a_42  * b_25;
      c_temp +=  a_43  * b_35;
      c_temp +=  a_44  * b_45;
      c_temp +=  a_45  * b_55;
      c[n + N*34 ] = c_temp;
      c_temp =  a_50  * b_00;
      c_temp +=  a_51  * b_10;
      c_temp +=  a_52  * b_20;
      c_temp +=  a_53  * b_30;
      c_temp +=  a_54  * b_40;
      c_temp +=  a_55  * b_50;
      c[n + N*5 ] = c_temp;
      c_temp =  a_50  * b_01;
      c_temp +=  a_51  * b_11;
      c_temp +=  a_52  * b_21;
      c_temp +=  a_53  * b_31;
      c_temp +=  a_54  * b_41;
      c_temp +=  a_55  * b_51;
      c[n + N*11 ] = c_temp;
      c_temp =  a_50  * b_02;
      c_temp +=  a_51  * b_12;
      c_temp +=  a_52  * b_22;
      c_temp +=  a_53  * b_32;
      c_temp +=  a_54  * b_42;
      c_temp +=  a_55  * b_52;
      c[n + N*17 ] = c_temp;
      c_temp =  a_50  * b_03;
      c_temp +=  a_51  * b_13;
      c_temp +=  a_52  * b_23;
      c_temp +=  a_53  * b_33;
      c_temp +=  a_54  * b_43;
      c_temp +=  a_55  * b_53;
      c[n + N*23 ] = c_temp;
      c_temp =  a_50  * b_04;
      c_temp +=  a_51  * b_14;
      c_temp +=  a_52  * b_24;
      c_temp +=  a_53  * b_34;
      c_temp +=  a_54  * b_44;
      c_temp +=  a_55  * b_54;
      c[n + N*29 ] = c_temp;
      c_temp =  a_50  * b_05;
      c_temp +=  a_51  * b_15;
      c_temp +=  a_52  * b_25;
      c_temp +=  a_53  * b_35;
      c_temp +=  a_54  * b_45;
      c_temp +=  a_55  * b_55;
      c[n + N*35 ] = c_temp;

    }//n = threadIdx.x + blockIdx.x * blockDim.x
  }//oLoop< nN; ++oLoop){
}

__global__ void raw_reg_c_mult_loop_unroll_const_kn(const float* RESTRICT const a, const float* RESTRICT const b, 
    float* c, const int N, const int nN)
{
  constexpr int NN = Nwidth;

  for (int oLoop = 0; oLoop< nN; ++oLoop){
    for (int n = threadIdx.x + blockIdx.x * blockDim.x;
         n < N;
         n += blockDim.x * gridDim.x) {
      float c_temp;
      float a_00 = a[n + NN*0];
      float b_00 = b[n + NN*0];
      c_temp =  a_00  * b_00;
      float a_01 = a[n + NN*6];
      float b_01 = b[n + NN*6];
      float a_10 = a[n + NN*1];
      float b_10 = b[n + NN*1];
      c_temp +=  a_01  * b_10;
      float a_02 = a[n + NN*12];
      float b_02 = b[n + NN*12];
      float a_20 = a[n + NN*2];
      float b_20 = b[n + NN*2];
      c_temp +=  a_02  * b_20;
      float a_03 = a[n + NN*18];
      float b_03 = b[n + NN*18];
      float a_30 = a[n + NN*3];
      float b_30 = b[n + NN*3];
      c_temp +=  a_03  * b_30;
      float a_04 = a[n + NN*24];
      float b_04 = b[n + NN*24];
      float a_40 = a[n + NN*4];
      float b_40 = b[n + NN*4];
      c_temp +=  a_04  * b_40;
      float a_05 = a[n + NN*30];
      float b_05 = b[n + NN*30];
      float a_50 = a[n + NN*5];
      float b_50 = b[n + NN*5];
      c_temp +=  a_05  * b_50;
      c[n + NN*0 ] = c_temp;
      c_temp =  a_00  * b_01;
      float a_11 = a[n + NN*7];
      float b_11 = b[n + NN*7];
      c_temp +=  a_01  * b_11;
      float a_21 = a[n + NN*8];
      float b_21 = b[n + NN*8];
      c_temp +=  a_02  * b_21;
      float a_31 = a[n + NN*9];
      float b_31 = b[n + NN*9];
      c_temp +=  a_03  * b_31;
      float a_41 = a[n + NN*10];
      float b_41 = b[n + NN*10];
      c_temp +=  a_04  * b_41;
      float a_51 = a[n + NN*11];
      float b_51 = b[n + NN*11];
      c_temp +=  a_05  * b_51;
      c[n + NN*6 ] = c_temp;
      c_temp =  a_00  * b_02;
      float a_12 = a[n + NN*13];
      float b_12 = b[n + NN*13];
      c_temp +=  a_01  * b_12;
      float a_22 = a[n + NN*14];
      float b_22 = b[n + NN*14];
      c_temp +=  a_02  * b_22;
      float a_32 = a[n + NN*15];
      float b_32 = b[n + NN*15];
      c_temp +=  a_03  * b_32;
      float a_42 = a[n + NN*16];
      float b_42 = b[n + NN*16];
      c_temp +=  a_04  * b_42;
      float a_52 = a[n + NN*17];
      float b_52 = b[n + NN*17];
      c_temp +=  a_05  * b_52;
      c[n + NN*12 ] = c_temp;
      c_temp =  a_00  * b_03;
      float a_13 = a[n + NN*19];
      float b_13 = b[n + NN*19];
      c_temp +=  a_01  * b_13;
      float a_23 = a[n + NN*20];
      float b_23 = b[n + NN*20];
      c_temp +=  a_02  * b_23;
      float a_33 = a[n + NN*21];
      float b_33 = b[n + NN*21];
      c_temp +=  a_03  * b_33;
      float a_43 = a[n + NN*22];
      float b_43 = b[n + NN*22];
      c_temp +=  a_04  * b_43;
      float a_53 = a[n + NN*23];
      float b_53 = b[n + NN*23];
      c_temp +=  a_05  * b_53;
      c[n + NN*18 ] = c_temp;
      c_temp =  a_00  * b_04;
      float a_14 = a[n + NN*25];
      float b_14 = b[n + NN*25];
      c_temp +=  a_01  * b_14;
      float a_24 = a[n + NN*26];
      float b_24 = b[n + NN*26];
      c_temp +=  a_02  * b_24;
      float a_34 = a[n + NN*27];
      float b_34 = b[n + NN*27];
      c_temp +=  a_03  * b_34;
      float a_44 = a[n + NN*28];
      float b_44 = b[n + NN*28];
      c_temp +=  a_04  * b_44;
      float a_54 = a[n + NN*29];
      float b_54 = b[n + NN*29];
      c_temp +=  a_05  * b_54;
      c[n + NN*24 ] = c_temp;
      c_temp =  a_00  * b_05;
      float a_15 = a[n + NN*31];
      float b_15 = b[n + NN*31];
      c_temp +=  a_01  * b_15;
      float a_25 = a[n + NN*32];
      float b_25 = b[n + NN*32];
      c_temp +=  a_02  * b_25;
      float a_35 = a[n + NN*33];
      float b_35 = b[n + NN*33];
      c_temp +=  a_03  * b_35;
      float a_45 = a[n + NN*34];
      float b_45 = b[n + NN*34];
      c_temp +=  a_04  * b_45;
      float a_55 = a[n + NN*35];
      float b_55 = b[n + NN*35];
      c_temp +=  a_05  * b_55;
      c[n + NN*30 ] = c_temp;
      c_temp =  a_10  * b_00;
      c_temp +=  a_11  * b_10;
      c_temp +=  a_12  * b_20;
      c_temp +=  a_13  * b_30;
      c_temp +=  a_14  * b_40;
      c_temp +=  a_15  * b_50;
      c[n + NN*1 ] = c_temp;
      c_temp =  a_10  * b_01;
      c_temp +=  a_11  * b_11;
      c_temp +=  a_12  * b_21;
      c_temp +=  a_13  * b_31;
      c_temp +=  a_14  * b_41;
      c_temp +=  a_15  * b_51;
      c[n + NN*7 ] = c_temp;
      c_temp =  a_10  * b_02;
      c_temp +=  a_11  * b_12;
      c_temp +=  a_12  * b_22;
      c_temp +=  a_13  * b_32;
      c_temp +=  a_14  * b_42;
      c_temp +=  a_15  * b_52;
      c[n + NN*13 ] = c_temp;
      c_temp =  a_10  * b_03;
      c_temp +=  a_11  * b_13;
      c_temp +=  a_12  * b_23;
      c_temp +=  a_13  * b_33;
      c_temp +=  a_14  * b_43;
      c_temp +=  a_15  * b_53;
      c[n + NN*19 ] = c_temp;
      c_temp =  a_10  * b_04;
      c_temp +=  a_11  * b_14;
      c_temp +=  a_12  * b_24;
      c_temp +=  a_13  * b_34;
      c_temp +=  a_14  * b_44;
      c_temp +=  a_15  * b_54;
      c[n + NN*25 ] = c_temp;
      c_temp =  a_10  * b_05;
      c_temp +=  a_11  * b_15;
      c_temp +=  a_12  * b_25;
      c_temp +=  a_13  * b_35;
      c_temp +=  a_14  * b_45;
      c_temp +=  a_15  * b_55;
      c[n + NN*31 ] = c_temp;
      c_temp =  a_20  * b_00;
      c_temp +=  a_21  * b_10;
      c_temp +=  a_22  * b_20;
      c_temp +=  a_23  * b_30;
      c_temp +=  a_24  * b_40;
      c_temp +=  a_25  * b_50;
      c[n + NN*2 ] = c_temp;
      c_temp =  a_20  * b_01;
      c_temp +=  a_21  * b_11;
      c_temp +=  a_22  * b_21;
      c_temp +=  a_23  * b_31;
      c_temp +=  a_24  * b_41;
      c_temp +=  a_25  * b_51;
      c[n + NN*8 ] = c_temp;
      c_temp =  a_20  * b_02;
      c_temp +=  a_21  * b_12;
      c_temp +=  a_22  * b_22;
      c_temp +=  a_23  * b_32;
      c_temp +=  a_24  * b_42;
      c_temp +=  a_25  * b_52;
      c[n + NN*14 ] = c_temp;
      c_temp =  a_20  * b_03;
      c_temp +=  a_21  * b_13;
      c_temp +=  a_22  * b_23;
      c_temp +=  a_23  * b_33;
      c_temp +=  a_24  * b_43;
      c_temp +=  a_25  * b_53;
      c[n + NN*20 ] = c_temp;
      c_temp =  a_20  * b_04;
      c_temp +=  a_21  * b_14;
      c_temp +=  a_22  * b_24;
      c_temp +=  a_23  * b_34;
      c_temp +=  a_24  * b_44;
      c_temp +=  a_25  * b_54;
      c[n + NN*26 ] = c_temp;
      c_temp =  a_20  * b_05;
      c_temp +=  a_21  * b_15;
      c_temp +=  a_22  * b_25;
      c_temp +=  a_23  * b_35;
      c_temp +=  a_24  * b_45;
      c_temp +=  a_25  * b_55;
      c[n + NN*32 ] = c_temp;
      c_temp =  a_30  * b_00;
      c_temp +=  a_31  * b_10;
      c_temp +=  a_32  * b_20;
      c_temp +=  a_33  * b_30;
      c_temp +=  a_34  * b_40;
      c_temp +=  a_35  * b_50;
      c[n + NN*3 ] = c_temp;
      c_temp =  a_30  * b_01;
      c_temp +=  a_31  * b_11;
      c_temp +=  a_32  * b_21;
      c_temp +=  a_33  * b_31;
      c_temp +=  a_34  * b_41;
      c_temp +=  a_35  * b_51;
      c[n + NN*9 ] = c_temp;
      c_temp =  a_30  * b_02;
      c_temp +=  a_31  * b_12;
      c_temp +=  a_32  * b_22;
      c_temp +=  a_33  * b_32;
      c_temp +=  a_34  * b_42;
      c_temp +=  a_35  * b_52;
      c[n + NN*15 ] = c_temp;
      c_temp =  a_30  * b_03;
      c_temp +=  a_31  * b_13;
      c_temp +=  a_32  * b_23;
      c_temp +=  a_33  * b_33;
      c_temp +=  a_34  * b_43;
      c_temp +=  a_35  * b_53;
      c[n + NN*21 ] = c_temp;
      c_temp =  a_30  * b_04;
      c_temp +=  a_31  * b_14;
      c_temp +=  a_32  * b_24;
      c_temp +=  a_33  * b_34;
      c_temp +=  a_34  * b_44;
      c_temp +=  a_35  * b_54;
      c[n + NN*27 ] = c_temp;
      c_temp =  a_30  * b_05;
      c_temp +=  a_31  * b_15;
      c_temp +=  a_32  * b_25;
      c_temp +=  a_33  * b_35;
      c_temp +=  a_34  * b_45;
      c_temp +=  a_35  * b_55;
      c[n + NN*33 ] = c_temp;
      c_temp =  a_40  * b_00;
      c_temp +=  a_41  * b_10;
      c_temp +=  a_42  * b_20;
      c_temp +=  a_43  * b_30;
      c_temp +=  a_44  * b_40;
      c_temp +=  a_45  * b_50;
      c[n + NN*4 ] = c_temp;
      c_temp =  a_40  * b_01;
      c_temp +=  a_41  * b_11;
      c_temp +=  a_42  * b_21;
      c_temp +=  a_43  * b_31;
      c_temp +=  a_44  * b_41;
      c_temp +=  a_45  * b_51;
      c[n + NN*10 ] = c_temp;
      c_temp =  a_40  * b_02;
      c_temp +=  a_41  * b_12;
      c_temp +=  a_42  * b_22;
      c_temp +=  a_43  * b_32;
      c_temp +=  a_44  * b_42;
      c_temp +=  a_45  * b_52;
      c[n + NN*16 ] = c_temp;
      c_temp =  a_40  * b_03;
      c_temp +=  a_41  * b_13;
      c_temp +=  a_42  * b_23;
      c_temp +=  a_43  * b_33;
      c_temp +=  a_44  * b_43;
      c_temp +=  a_45  * b_53;
      c[n + NN*22 ] = c_temp;
      c_temp =  a_40  * b_04;
      c_temp +=  a_41  * b_14;
      c_temp +=  a_42  * b_24;
      c_temp +=  a_43  * b_34;
      c_temp +=  a_44  * b_44;
      c_temp +=  a_45  * b_54;
      c[n + NN*28 ] = c_temp;
      c_temp =  a_40  * b_05;
      c_temp +=  a_41  * b_15;
      c_temp +=  a_42  * b_25;
      c_temp +=  a_43  * b_35;
      c_temp +=  a_44  * b_45;
      c_temp +=  a_45  * b_55;
      c[n + NN*34 ] = c_temp;
      c_temp =  a_50  * b_00;
      c_temp +=  a_51  * b_10;
      c_temp +=  a_52  * b_20;
      c_temp +=  a_53  * b_30;
      c_temp +=  a_54  * b_40;
      c_temp +=  a_55  * b_50;
      c[n + NN*5 ] = c_temp;
      c_temp =  a_50  * b_01;
      c_temp +=  a_51  * b_11;
      c_temp +=  a_52  * b_21;
      c_temp +=  a_53  * b_31;
      c_temp +=  a_54  * b_41;
      c_temp +=  a_55  * b_51;
      c[n + NN*11 ] = c_temp;
      c_temp =  a_50  * b_02;
      c_temp +=  a_51  * b_12;
      c_temp +=  a_52  * b_22;
      c_temp +=  a_53  * b_32;
      c_temp +=  a_54  * b_42;
      c_temp +=  a_55  * b_52;
      c[n + NN*17 ] = c_temp;
      c_temp =  a_50  * b_03;
      c_temp +=  a_51  * b_13;
      c_temp +=  a_52  * b_23;
      c_temp +=  a_53  * b_33;
      c_temp +=  a_54  * b_43;
      c_temp +=  a_55  * b_53;
      c[n + NN*23 ] = c_temp;
      c_temp =  a_50  * b_04;
      c_temp +=  a_51  * b_14;
      c_temp +=  a_52  * b_24;
      c_temp +=  a_53  * b_34;
      c_temp +=  a_54  * b_44;
      c_temp +=  a_55  * b_54;
      c[n + NN*29 ] = c_temp;
      c_temp =  a_50  * b_05;
      c_temp +=  a_51  * b_15;
      c_temp +=  a_52  * b_25;
      c_temp +=  a_53  * b_35;
      c_temp +=  a_54  * b_45;
      c_temp +=  a_55  * b_55;
      c[n + NN*35 ] = c_temp;

    }//n = threadIdx.x + blockIdx.x * blockDim.x
  }//oLoop< nN; ++oLoop){
}


/** similar to raw_reg_c_mult_loop_unroll_const_kn but with different grouping of 
    loads and multiplication: all loads needed for the product of a row/column
    are loaded entirely and then all the multiplications/additions happen.

    The code only values from the memory to local variables (i.e. registers)
    if it has not done so for a previous product.
*/
__global__ void raw_reg_c_mult_loop_unroll2_const_kn(const float* RESTRICT const a, const float* RESTRICT const b, 
    float* c, const int N, const int nN)
{
  for (int oLoop = 0; oLoop< nN; ++oLoop){
    for (int n = threadIdx.x + blockIdx.x * blockDim.x;
         n < N;
         n += blockDim.x * gridDim.x) {

	 float c_temp;
	 float a_00 = a[n + N*0];
	 float b_00 = b[n + N*0];
	 float a_01 = a[n + N*6];
	 float b_01 = b[n + N*6];
	 float a_10 = a[n + N*1];
	 float b_10 = b[n + N*1];
	 float a_02 = a[n + N*12];
	 float b_02 = b[n + N*12];
	 float a_20 = a[n + N*2];
	 float b_20 = b[n + N*2];
	 float a_03 = a[n + N*18];
	 float b_03 = b[n + N*18];
	 float a_30 = a[n + N*3];
	 float b_30 = b[n + N*3];
	 float a_04 = a[n + N*24];
	 float b_04 = b[n + N*24];
	 float a_40 = a[n + N*4];
	 float b_40 = b[n + N*4];
	 float a_05 = a[n + N*30];
	 float b_05 = b[n + N*30];
	 float a_50 = a[n + N*5];
	 float b_50 = b[n + N*5];
	 c_temp =  a_00  * b_00;
	 c_temp +=  a_01  * b_10;
	 c_temp +=  a_02  * b_20;
	 c_temp +=  a_03  * b_30;
	 c_temp +=  a_04  * b_40;
	 c_temp +=  a_05  * b_50;
	 c[n + N*0 ] = c_temp;
	 float a_11 = a[n + N*7];
	 float b_11 = b[n + N*7];
	 float a_21 = a[n + N*8];
	 float b_21 = b[n + N*8];
	 float a_31 = a[n + N*9];
	 float b_31 = b[n + N*9];
	 float a_41 = a[n + N*10];
	 float b_41 = b[n + N*10];
	 float a_51 = a[n + N*11];
	 float b_51 = b[n + N*11];
	 c_temp =  a_00  * b_01;
	 c_temp +=  a_01  * b_11;
	 c_temp +=  a_02  * b_21;
	 c_temp +=  a_03  * b_31;
	 c_temp +=  a_04  * b_41;
	 c_temp +=  a_05  * b_51;
	 c[n + N*6 ] = c_temp;
	 float a_12 = a[n + N*13];
	 float b_12 = b[n + N*13];
	 float a_22 = a[n + N*14];
	 float b_22 = b[n + N*14];
	 float a_32 = a[n + N*15];
	 float b_32 = b[n + N*15];
	 float a_42 = a[n + N*16];
	 float b_42 = b[n + N*16];
	 float a_52 = a[n + N*17];
	 float b_52 = b[n + N*17];
	 c_temp =  a_00  * b_02;
	 c_temp +=  a_01  * b_12;
	 c_temp +=  a_02  * b_22;
	 c_temp +=  a_03  * b_32;
	 c_temp +=  a_04  * b_42;
	 c_temp +=  a_05  * b_52;
	 c[n + N*12 ] = c_temp;
	 float a_13 = a[n + N*19];
	 float b_13 = b[n + N*19];
	 float a_23 = a[n + N*20];
	 float b_23 = b[n + N*20];
	 float a_33 = a[n + N*21];
	 float b_33 = b[n + N*21];
	 float a_43 = a[n + N*22];
	 float b_43 = b[n + N*22];
	 float a_53 = a[n + N*23];
	 float b_53 = b[n + N*23];
	 c_temp =  a_00  * b_03;
	 c_temp +=  a_01  * b_13;
	 c_temp +=  a_02  * b_23;
	 c_temp +=  a_03  * b_33;
	 c_temp +=  a_04  * b_43;
	 c_temp +=  a_05  * b_53;
	 c[n + N*18 ] = c_temp;
	 float a_14 = a[n + N*25];
	 float b_14 = b[n + N*25];
	 float a_24 = a[n + N*26];
	 float b_24 = b[n + N*26];
	 float a_34 = a[n + N*27];
	 float b_34 = b[n + N*27];
	 float a_44 = a[n + N*28];
	 float b_44 = b[n + N*28];
	 float a_54 = a[n + N*29];
	 float b_54 = b[n + N*29];
	 c_temp =  a_00  * b_04;
	 c_temp +=  a_01  * b_14;
	 c_temp +=  a_02  * b_24;
	 c_temp +=  a_03  * b_34;
	 c_temp +=  a_04  * b_44;
	 c_temp +=  a_05  * b_54;
	 c[n + N*24 ] = c_temp;
	 float a_15 = a[n + N*31];
	 float b_15 = b[n + N*31];
	 float a_25 = a[n + N*32];
	 float b_25 = b[n + N*32];
	 float a_35 = a[n + N*33];
	 float b_35 = b[n + N*33];
	 float a_45 = a[n + N*34];
	 float b_45 = b[n + N*34];
	 float a_55 = a[n + N*35];
	 float b_55 = b[n + N*35];
	 c_temp =  a_00  * b_05;
	 c_temp +=  a_01  * b_15;
	 c_temp +=  a_02  * b_25;
	 c_temp +=  a_03  * b_35;
	 c_temp +=  a_04  * b_45;
	 c_temp +=  a_05  * b_55;
	 c[n + N*30 ] = c_temp;
	 c_temp =  a_10  * b_00;
	 c_temp +=  a_11  * b_10;
	 c_temp +=  a_12  * b_20;
	 c_temp +=  a_13  * b_30;
	 c_temp +=  a_14  * b_40;
	 c_temp +=  a_15  * b_50;
	 c[n + N*1 ] = c_temp;
	 c_temp =  a_10  * b_01;
	 c_temp +=  a_11  * b_11;
	 c_temp +=  a_12  * b_21;
	 c_temp +=  a_13  * b_31;
	 c_temp +=  a_14  * b_41;
	 c_temp +=  a_15  * b_51;
	 c[n + N*7 ] = c_temp;
	 c_temp =  a_10  * b_02;
	 c_temp +=  a_11  * b_12;
	 c_temp +=  a_12  * b_22;
	 c_temp +=  a_13  * b_32;
	 c_temp +=  a_14  * b_42;
	 c_temp +=  a_15  * b_52;
	 c[n + N*13 ] = c_temp;
	 c_temp =  a_10  * b_03;
	 c_temp +=  a_11  * b_13;
	 c_temp +=  a_12  * b_23;
	 c_temp +=  a_13  * b_33;
	 c_temp +=  a_14  * b_43;
	 c_temp +=  a_15  * b_53;
	 c[n + N*19 ] = c_temp;
	 c_temp =  a_10  * b_04;
	 c_temp +=  a_11  * b_14;
	 c_temp +=  a_12  * b_24;
	 c_temp +=  a_13  * b_34;
	 c_temp +=  a_14  * b_44;
	 c_temp +=  a_15  * b_54;
	 c[n + N*25 ] = c_temp;
	 c_temp =  a_10  * b_05;
	 c_temp +=  a_11  * b_15;
	 c_temp +=  a_12  * b_25;
	 c_temp +=  a_13  * b_35;
	 c_temp +=  a_14  * b_45;
	 c_temp +=  a_15  * b_55;
	 c[n + N*31 ] = c_temp;
	 c_temp =  a_20  * b_00;
	 c_temp +=  a_21  * b_10;
	 c_temp +=  a_22  * b_20;
	 c_temp +=  a_23  * b_30;
	 c_temp +=  a_24  * b_40;
	 c_temp +=  a_25  * b_50;
	 c[n + N*2 ] = c_temp;
	 c_temp =  a_20  * b_01;
	 c_temp +=  a_21  * b_11;
	 c_temp +=  a_22  * b_21;
	 c_temp +=  a_23  * b_31;
	 c_temp +=  a_24  * b_41;
	 c_temp +=  a_25  * b_51;
	 c[n + N*8 ] = c_temp;
	 c_temp =  a_20  * b_02;
	 c_temp +=  a_21  * b_12;
	 c_temp +=  a_22  * b_22;
	 c_temp +=  a_23  * b_32;
	 c_temp +=  a_24  * b_42;
	 c_temp +=  a_25  * b_52;
	 c[n + N*14 ] = c_temp;
	 c_temp =  a_20  * b_03;
	 c_temp +=  a_21  * b_13;
	 c_temp +=  a_22  * b_23;
	 c_temp +=  a_23  * b_33;
	 c_temp +=  a_24  * b_43;
	 c_temp +=  a_25  * b_53;
	 c[n + N*20 ] = c_temp;
	 c_temp =  a_20  * b_04;
	 c_temp +=  a_21  * b_14;
	 c_temp +=  a_22  * b_24;
	 c_temp +=  a_23  * b_34;
	 c_temp +=  a_24  * b_44;
	 c_temp +=  a_25  * b_54;
	 c[n + N*26 ] = c_temp;
	 c_temp =  a_20  * b_05;
	 c_temp +=  a_21  * b_15;
	 c_temp +=  a_22  * b_25;
	 c_temp +=  a_23  * b_35;
	 c_temp +=  a_24  * b_45;
	 c_temp +=  a_25  * b_55;
	 c[n + N*32 ] = c_temp;
	 c_temp =  a_30  * b_00;
	 c_temp +=  a_31  * b_10;
	 c_temp +=  a_32  * b_20;
	 c_temp +=  a_33  * b_30;
	 c_temp +=  a_34  * b_40;
	 c_temp +=  a_35  * b_50;
	 c[n + N*3 ] = c_temp;
	 c_temp =  a_30  * b_01;
	 c_temp +=  a_31  * b_11;
	 c_temp +=  a_32  * b_21;
	 c_temp +=  a_33  * b_31;
	 c_temp +=  a_34  * b_41;
	 c_temp +=  a_35  * b_51;
	 c[n + N*9 ] = c_temp;
	 c_temp =  a_30  * b_02;
	 c_temp +=  a_31  * b_12;
	 c_temp +=  a_32  * b_22;
	 c_temp +=  a_33  * b_32;
	 c_temp +=  a_34  * b_42;
	 c_temp +=  a_35  * b_52;
	 c[n + N*15 ] = c_temp;
	 c_temp =  a_30  * b_03;
	 c_temp +=  a_31  * b_13;
	 c_temp +=  a_32  * b_23;
	 c_temp +=  a_33  * b_33;
	 c_temp +=  a_34  * b_43;
	 c_temp +=  a_35  * b_53;
	 c[n + N*21 ] = c_temp;
	 c_temp =  a_30  * b_04;
	 c_temp +=  a_31  * b_14;
	 c_temp +=  a_32  * b_24;
	 c_temp +=  a_33  * b_34;
	 c_temp +=  a_34  * b_44;
	 c_temp +=  a_35  * b_54;
	 c[n + N*27 ] = c_temp;
	 c_temp =  a_30  * b_05;
	 c_temp +=  a_31  * b_15;
	 c_temp +=  a_32  * b_25;
	 c_temp +=  a_33  * b_35;
	 c_temp +=  a_34  * b_45;
	 c_temp +=  a_35  * b_55;
	 c[n + N*33 ] = c_temp;
	 c_temp =  a_40  * b_00;
	 c_temp +=  a_41  * b_10;
	 c_temp +=  a_42  * b_20;
	 c_temp +=  a_43  * b_30;
	 c_temp +=  a_44  * b_40;
	 c_temp +=  a_45  * b_50;
	 c[n + N*4 ] = c_temp;
	 c_temp =  a_40  * b_01;
	 c_temp +=  a_41  * b_11;
	 c_temp +=  a_42  * b_21;
	 c_temp +=  a_43  * b_31;
	 c_temp +=  a_44  * b_41;
	 c_temp +=  a_45  * b_51;
	 c[n + N*10 ] = c_temp;
	 c_temp =  a_40  * b_02;
	 c_temp +=  a_41  * b_12;
	 c_temp +=  a_42  * b_22;
	 c_temp +=  a_43  * b_32;
	 c_temp +=  a_44  * b_42;
	 c_temp +=  a_45  * b_52;
	 c[n + N*16 ] = c_temp;
	 c_temp =  a_40  * b_03;
	 c_temp +=  a_41  * b_13;
	 c_temp +=  a_42  * b_23;
	 c_temp +=  a_43  * b_33;
	 c_temp +=  a_44  * b_43;
	 c_temp +=  a_45  * b_53;
	 c[n + N*22 ] = c_temp;
	 c_temp =  a_40  * b_04;
	 c_temp +=  a_41  * b_14;
	 c_temp +=  a_42  * b_24;
	 c_temp +=  a_43  * b_34;
	 c_temp +=  a_44  * b_44;
	 c_temp +=  a_45  * b_54;
	 c[n + N*28 ] = c_temp;
	 c_temp =  a_40  * b_05;
	 c_temp +=  a_41  * b_15;
	 c_temp +=  a_42  * b_25;
	 c_temp +=  a_43  * b_35;
	 c_temp +=  a_44  * b_45;
	 c_temp +=  a_45  * b_55;
	 c[n + N*34 ] = c_temp;
	 c_temp =  a_50  * b_00;
	 c_temp +=  a_51  * b_10;
	 c_temp +=  a_52  * b_20;
	 c_temp +=  a_53  * b_30;
	 c_temp +=  a_54  * b_40;
	 c_temp +=  a_55  * b_50;
	 c[n + N*5 ] = c_temp;
	 c_temp =  a_50  * b_01;
	 c_temp +=  a_51  * b_11;
	 c_temp +=  a_52  * b_21;
	 c_temp +=  a_53  * b_31;
	 c_temp +=  a_54  * b_41;
	 c_temp +=  a_55  * b_51;
	 c[n + N*11 ] = c_temp;
	 c_temp =  a_50  * b_02;
	 c_temp +=  a_51  * b_12;
	 c_temp +=  a_52  * b_22;
	 c_temp +=  a_53  * b_32;
	 c_temp +=  a_54  * b_42;
	 c_temp +=  a_55  * b_52;
	 c[n + N*17 ] = c_temp;
	 c_temp =  a_50  * b_03;
	 c_temp +=  a_51  * b_13;
	 c_temp +=  a_52  * b_23;
	 c_temp +=  a_53  * b_33;
	 c_temp +=  a_54  * b_43;
	 c_temp +=  a_55  * b_53;
	 c[n + N*23 ] = c_temp;
	 c_temp =  a_50  * b_04;
	 c_temp +=  a_51  * b_14;
	 c_temp +=  a_52  * b_24;
	 c_temp +=  a_53  * b_34;
	 c_temp +=  a_54  * b_44;
	 c_temp +=  a_55  * b_54;
	 c[n + N*29 ] = c_temp;
	 c_temp =  a_50  * b_05;
	 c_temp +=  a_51  * b_15;
	 c_temp +=  a_52  * b_25;
	 c_temp +=  a_53  * b_35;
	 c_temp +=  a_54  * b_45;
	 c_temp +=  a_55  * b_55;
	 c[n + N*35 ] = c_temp;
    }//n = threadIdx.x + blockIdx.x * blockDim.x
  }//oLoop< nN; ++oLoop){
}

__global__ void raw_shared_mult_kn(const float* RESTRICT a, const float* RESTRICT b, float* c, const int N)
{
  for (int n = threadIdx.x + blockIdx.x * blockDim.x;
       n < N;
       n += blockDim.x * gridDim.x) {

    int tix = threadIdx.x;

    __shared__ float sh_a[36][block_size];
    __shared__ float sh_b[36][block_size];

    for (int i = 0; i < 36; ++i) {
      sh_a[i][tix] = a[n + N*i];
    }
    for (int i = 0; i < 36; ++i) {
      sh_b[i][tix] = b[n + N*i];
    }
    __syncthreads();

    for (int i = 0; i < 6; ++i) {
      for (int j = 0; j < 6; ++j) {
        float c_tmp = 0;
        for (int k = 0; k < 6; ++k) {
          /*c_tmp += a(n, i, k) * b(n, k, j);*/
          c_tmp += sh_a[k + 6* i][tix] 
            * sh_b[j + 6 * k][tix];
          /*c_tmp += sh_a[0][tix] ;*/
            /** sh_b[j + GPlexMP::kCols * k][tix];*/
        }
        c[n + N*(i + 6*j)] = c_tmp;
      }
    }
  }
}

__global__ void raw_reg_mult_kn(const float* RESTRICT a, const float* RESTRICT b, float* c, const int N)
{
  for (int n = threadIdx.x + blockIdx.x * blockDim.x;
      n < N;
      n += blockDim.x * gridDim.x) {

    float reg_a[36];
    float reg_b[36];

    for (int i = 0; i < 36; ++i) {
      reg_a[i] = a[n + N*i];
    }
    for (int i = 0; i < 36; ++i) {
      reg_b[i] = b[n + N*i];
    }

    for (int i = 0; i < 6; ++i) {
      for (int j = 0; j < 6; ++j) {
        float c_tmp = 0;
        for (int k = 0; k < 6; ++k) {
          c_tmp += reg_a[i+6*k] * reg_b[k+6+j];
        }
        c[n + N*(i + 6*j)] = c_tmp;
      }
    }
  }
}

__global__ void set_mem(float* a, float val, size_t N) {

  for (int n = threadIdx.x + blockIdx.x * blockDim.x;
       n < N;
       n += blockDim.x * gridDim.x) {

     for (int j = 0; j < 36; j++) {
       a[36*n + j] = val;
     }
  }
}

bool check(int N, float* c, float *h, bool managed)
{
  const float eps = 1e-30;
  int n = 36*N;
  float c0, c36;
  if (managed) {
    c0 = c[0];
    c36 = c[36];
    int device = -1;
    hipGetDevice(&device);
    hipMemPrefetchAsync(c, n*sizeof(float), device, NULL);
  } else {
    hipMemcpy(h, c, n*sizeof(float), hipMemcpyDeviceToHost);
    c0 = h[0];
    c36 = h[36];
  }
  bool pass = (std::abs(c0 - c36) < eps) && (std::abs(c0 - 6.0f) < eps);
  if (!pass) {
    std::cout << "Fail check c[0]=" << c0 << " c[36]=" << c36 << std::endl;
  }
  return pass;
}

void raw_run_naive_mul(int iter, bool managed)
{
  constexpr int N = Nwidth;
  constexpr int sz = 36*N*sizeof(float);

  float* a;
  float* b;
  float* c;
  float h[36*N];

  if (managed) {
    hipMallocManaged((void**)&a, sz);
    hipMallocManaged((void**)&b, sz);
    hipMallocManaged((void**)&c, sz);

    int device = -1;
    hipGetDevice(&device);
    hipMemPrefetchAsync(a, sz, device, NULL);
    hipMemPrefetchAsync(b, sz, device, NULL);
    hipMemPrefetchAsync(c, sz, device, NULL);
  } else {
    hipMalloc((void**)&a, sz);
    hipMalloc((void**)&b, sz);
    hipMalloc((void**)&c, sz);
  }
  cudaCheckErrorSync();

  dim3 grid (((N-1)/block_size + 1), 1, 1);
  dim3 block (block_size, 1, 1);

  set_mem <<< grid, block >>> (a, 1.f , N);
  set_mem <<< grid, block >>> (b, 1.f, N);
  set_mem <<< grid, block >>> (c, 0.f, N);

  cudaCheckErrorSync();

  if (managed) {
    hipMemAdvise(a, sz, hipMemAdviseSetReadMostly, 0);
    hipMemAdvise(b, sz, hipMemAdviseSetReadMostly, 0);
  }

  for (int i = 0; i < iter; ++i) {
    set_mem <<< grid, block >>> (c, 0.f, N);
    raw_naive_mult_kn <<< grid, block >>> (a, b, c, N);
  }
  cudaCheckErrorSync();
  assert(check(N, c, h, managed));

  for (int i = 0; i < iter; ++i)
    raw_reg_c_mult_kn <<< grid, block >>> (a, b, c, N);
  cudaCheckErrorSync();
  assert(check(N, c, h, managed));

  for (int i = 0; i < iter; ++i)
    raw_shared_mult_kn <<< grid, block >>> (a, b, c, N);
  cudaCheckErrorSync();
  assert(check(N, c, h, managed));

  for (int i = 0; i < iter; ++i)
    raw_reg_mult_kn <<< grid, block >>> (a, b, c, N);
  cudaCheckErrorSync();
  assert(check(N, c, h, managed));

  const float alpha = 1.0f;
  const float beta = 0.0f;
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  // cublas gives an address error with managed memory and
  // large # of arrays.
  if (!managed) {
    for (int i = 0; i < iter; ++i)
      hipblasSgemmStridedBatched(handle,
                                HIPBLAS_OP_N, 
                                HIPBLAS_OP_N,
                                6, 6, 6,
                                &alpha,
                                a, 6,
                                36*sizeof(float),
                                b, 6,
                                36*sizeof(float),
                                &beta,
                                c, 6, 
                                36*sizeof(float), 
                                N);
    cudaCheckErrorSync();
    assert(check(N, c, h, managed));
  }

  raw_regall_c_mult_loop_kn <<< grid, block >>> (a, b, c, N, iter);
  cudaCheckErrorSync();
  //no checks here because the operations are not the same

  raw_reg_c_mult_loop_kn <<< grid, block >>> (a, b, c, N, iter);
  cudaCheckErrorSync();
  assert(check(N, c, h, managed));

  raw_reg_c_mult_loop_kn_vl <<< grid, block >>> (a, b, c, N, iter);
  cudaCheckErrorSync();
  assert(check(N, c, h, managed));

  raw_reg_c_mult_loop_kn_vl_vs <<< grid, block >>> (a, b, c, N, iter);
  cudaCheckErrorSync();
  assert(check(N, c, h, managed));

  raw_reg_c_mult_loop_kn_vl_transp <<< grid, block >>> (a, b, c, N, iter);
  cudaCheckErrorSync();
  // since this kernel assumes a transposed layout
  // the check may fail in the future
  assert(check(N, c, h, managed));

  raw_reg_c_mult_loop_unroll_kn <<< grid, block >>> (a, b, c, N, iter);
  cudaCheckErrorSync();
  assert(check(N, c, h, managed));

  raw_reg_c_mult_loop_unroll_const_kn <<< grid, block >>> (a, b, c, N, iter);
  cudaCheckErrorSync();
  assert(check(N, c, h, managed));

  hipFree(a);
  hipFree(b);
  hipFree(c);
  cudaCheckErrorSync();
}
